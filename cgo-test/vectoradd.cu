#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}


extern "C" {
	void vectoradd(float *a, float *b, float *out, int N) {
		float *d_a, *d_b, *d_out;

		hipMalloc((void**)&d_a, sizeof(float) * N);
		hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);

		hipMalloc((void**)&d_b, sizeof(float) * N);
		hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

		hipMalloc((void**)&d_out, sizeof(float) * N);

		// Main function
		vector_add<<<1,1>>>(d_out, d_a, d_b, N);

		hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_out);
	}
}
